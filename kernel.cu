#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// variáveis globais
// Legenda:
// int a e int b são vetores quaisquer
// int c é a soma deles
int* a, int* b, int* c;

//kernel -- GPU
__global__ void somaVec(int N, int* a, int* b, int* c) 
{
	int i = threadIdx.x;
	// identificando a thread 
	while (i < N) {
		c[i] = a[i] + b[i];
		i += blockDim.x;
	}
	
}

// código GPU
int main() {
	hipDeviceReset();
	int* g_a, int* g_b, int* g_c;

	int n = 1024;
	int tamanho = n * sizeof(int);

	// malloc das variáveis a,b e c. liberar espaço na memória
	a = (int*)malloc(tamanho);
	b = (int*)malloc(tamanho);
	c = (int*)malloc(tamanho);

	// malloc da GPU
	hipMalloc((void**)&g_a, tamanho);
	hipMalloc((void**)&g_b, tamanho);
	hipMalloc((void**)&g_c, tamanho);

	// controle dos valores
	for (int i = 0; i < n; i++)
		a[i] = i, b[i] = i;

	//transferindo os dados do host para o device/gpu
	hipMemcpy(g_a, a, tamanho, hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, tamanho, hipMemcpyHostToDevice);

	// execução do kernel
	somaVec <<< 1, 1024 >>> (n, g_a, g_b, g_c);

	hipDeviceSynchronize();

	// devolendo para o host
	hipMemcpy(c, g_c, tamanho, hipMemcpyDeviceToHost);

	printf(" \n Resultado da Soma: \n");
	for (int i = 0; i < n; i++) {
			printf("\n");
			printf("%d", c[i]);
		}

	// liberando memória
	hipFree(g_a);
	hipFree(g_b);
	hipFree(g_c);
	return 0;

}